// System includes
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <ctime>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

#include <>
//#include <hip/device_functions.h>

// Custom header 
#include "AES_final.h"
//
#include "128-cmac.cuh"

int main() {
	hipSetDevice(0);
	CMAC128ExhaustiveSearch();
	return 0;
}
